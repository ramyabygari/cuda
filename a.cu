#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
__global__void addint(int *a,int *b)
{
a[0]=a[0]+b[0];
}
int main()
{
int a=5;
int b=9;
int *d_a,*d_b;
hipMalloc(&d_a,sizeof(int));
hipMalloc(&d_b,sizeof(int));
hipMemcpy(d_a,&a,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_b,&b,sizeof(int),hipMemcpyHostToDevice);
addint<<<1,1>>>(d_a,d_b);
hipMemcpy(&a,d_a,sizeof(int),hipMemcpyDeviceToHost);
cout<a;
return 0;
}
