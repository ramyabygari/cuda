#include "hip/hip_runtime.h"
// The dataset generator generates all the datasets into one single pair of input files.

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <math.h>

using namespace std;

float truncs(float n)
{
  float nearest = roundf(n * 100) / 100;
  return nearest;
}


int main(int argc, char *argv[]) {

  if(argc < 4)
  {
    cout<<"Ensure that an output file and 2 input files are passed in as arguments when running this program\n";
    exit(0);
  }

  /* parse the input arguments */
  //@@ Insert code here

  char file1[100], file2[100], file3[100],file4[100];
  strcpy(file3,argv[1]);
  strcpy(file1,argv[2]);
  strcpy(file2,argv[3]);
  if(argc > 4)
    strcpy(file4,argv[4]);

  
  FILE *handle1 = fopen(file1, "r");
  FILE *handle2 = fopen(file2, "r");
  FILE *handle3 = fopen(file3,"r");
  FILE *handle4;

  if(argc > 4) //To write into optional output file
  {
     handle4 = fopen(file4, "w");
  }

  for(int i = 0;i < 10; i++)
{
  float *hostInput1 = NULL;
  float *hostInput2 = NULL;
  float *hostOutput = NULL;
  int inputLength;

  //Read size of vector
  fscanf(handle1, "%d", &inputLength);
  fscanf(handle2, "%d", &inputLength);
  fscanf(handle3, "%d", &inputLength);

  hostInput1 = (float*) malloc(inputLength*sizeof(float));
  hostInput2 = (float*) malloc(inputLength*sizeof(float));

  
  // Import host input data
  //@@ Read data from the raw files here
  //@@ Insert code here
  for (int ii = 0; ii < inputLength; ii++) {
      fscanf(handle1, "%f", &hostInput1[ii]);
      fscanf(handle2, "%f",&hostInput2[ii]);
  }

  // Declare and allocate host output
  //@@ Insert code here
  hostOutput = (float*) malloc(inputLength*sizeof(float));

  // Declare and allocate thrust device input and output vectors
  //@@ Insert code here
  thrust::device_vector<float> da(hostInput1,hostInput1+inputLength); 
  thrust::device_vector<float> db(hostInput2,hostInput2+inputLength); 
  thrust::device_vector<float> dc(hostOutput,hostOutput+inputLength); 

  // Copy to device
  //@@ Insert code here

  // Execute vector addition
  //@@ Insert Code here
  thrust::transform(da.begin(), da.end(), db.begin(), dc.begin(), thrust::plus<float>());
  /////////////////////////////////////////////////////////

  // Copy data back to host
  //@@ Insert code here
  thrust::copy(dc.begin(), dc.end(), hostOutput);



  // Verifying results
  if(argc>4)
    fprintf(handle4, "%d", inputLength);
  int flag = 1;
  for(int j = 0; j < inputLength; j++)
    {
      float n;
      fscanf(handle3,"%f",&n);
      if(flag)
      {
          if(truncs(n) != truncs(hostOutput[j]))
        {
          cout<<"Dataset "<<i<<" could not be verified\n";
          //cout<<truncs(n)<<" "<<truncs(hostOutput[j])<<" "<<j<<endl;
          flag = 0;
        }
      }
      //hostOutput[j] = truncs(hostOutput[j]);
      if(argc>4)
        fprintf(handle4, "\n%f", hostOutput[j]);
      
    }

    if(flag)
      cout<<"Dataset "<<i<<" verified\n";

  cout<<endl;



  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  }

  if(argc > 4)
    cout<<"Output written into file: "<<file4<<endl;

  fclose(handle1);
  fclose(handle2);
  
  return 0;
}
