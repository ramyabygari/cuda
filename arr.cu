#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<ctime>

using namespace std;

__global__ void addarr(int *d_a,int *d_b)
{
int id=threadIdx.x+(blockIdx.x*blockDim.x)

if(id<count)
{
d_a[id]=d_a[id]+d_b[id];
}
}


int main()
{
srand(time(NULL));
int count=100;
int *h_a=new int[count];
int *h_b=new int[count];
for(int i=0;i<count;i++){
h_a[i]=rand()%1000;
h_b[i]=rand()%1000;}
cout<<"Before addition\n";
for(int i=0;i<5;i++)
cout<<h_a[i]<<" "<<h_b[i];

int *d_a,d_b;
hipMalloc(&d_a,sizeof(int)*count);
hipMalloc(&d_b,sizeof(int)*count);
hipMemcpy(d_a,&h_a,sizeof(int)*count,hipMemcpyHostToDevice);
hipMemcpy(d_b,&h_b,sizeof(int)*count,hipMemcpyHostToDevice);
addarr<<<count/256+1,256>>>(d_a,d_b,count);
hipMemcpy(&h_a,d_a,sizeof(int)*count,hipMemcpyDeviceToHost);
for(i=0;i<5;i++)
cout<h_a[i]<<" ";

}
